#include "hip/hip_runtime.h"
#include "Simulator.hpp"
#include "Particle.hpp"
#include "Constants.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

__host__ ParallelSimulator::ParallelSimulator(const Scene& scene) {
    _n = scene.particles.size();
    hipMalloc(&n, sizeof(size_t));
    hipMemset(&n, _n, sizeof(size_t));   
    hipMalloc(&threads, sizeof(size_t));
    hipMemset(&threads, Constants::threads_per_block, sizeof(size_t));
    _blocks = (_n + Constants::threads_per_block - 1) / Constants::threads_per_block;
    hipMalloc(&blocks, sizeof(size_t));
    hipMemset(&blocks, _blocks, sizeof(size_t));

    hipMalloc(&bbox_mins, sizeof(glm::dvec3));
    hipMalloc(&bbox_maxs, sizeof(glm::dvec3));
    hipMemcpy(bbox_mins, &scene.bbox_mins, sizeof(glm::dvec3), hipMemcpyHostToDevice);
    hipMemcpy(bbox_maxs, &scene.bbox_maxs, sizeof(glm::dvec3), hipMemcpyHostToDevice);

    hipMalloc(&grid_width, sizeof(size_t));
    hipMalloc(&grid_height, sizeof(size_t));
    hipMalloc(&grid_length, sizeof(size_t));
    size_t width = (scene.bbox_maxs.x - scene.bbox_mins.x) / Constants::h + 1;
    size_t height = (scene.bbox_maxs.y - scene.bbox_mins.y) / Constants::h + 1;
    size_t length = (scene.bbox_maxs.z - scene.bbox_mins.z) / Constants::h + 1;
    hipMemset(grid_width, width, sizeof(size_t));
    hipMemset(grid_height, height, sizeof(size_t));
    hipMemset(grid_length, length, sizeof(size_t));

    hipMalloc((void**)&lambdas, sizeof(double) * n);
    hipMalloc((void**)&densities, sizeof(double) * n);
    hipMalloc((void**)&particles, sizeof(Particle) * n);
    hipMalloc((void**)&sorted_particles, sizeof(Particle) * n);
    hipMalloc((void**)&neighbor_starts, sizeof(int) * n);

    _total_cells = width * height * length; // overflow?
    hipMalloc(&total_cells, sizeof(size_t));
    hipMemset(&total_cells, _total_cells, sizeof(size_t)); 
    hipMalloc((void**)&bins, sizeof(int) * total_cells);
    hipMalloc((void**)&prefix_bins, sizeof(int) * total_cells);

    hipMalloc((void**)&delta_pos, sizeof(glm::dvec3) * n);
    hipMalloc((void**)&delta_vel, sizeof(glm::dvec3) * n);
}

__host__ ParallelSimulator::~ParallelSimulator() {
    hipFree(n);
    hipFree(threads);
    hipFree(blocks);
    hipFree(total_cells);
    hipFree(bbox_mins);
    hipFree(bbox_maxs);
    hipFree(grid_width);
    hipFree(grid_height);
    hipFree(grid_length);
    hipFree(lambdas);
    hipFree(densities);
    hipFree(particles);
    hipFree(sorted_particles);
    hipFree(bins);
    hipFree(prefix_bins);
    hipFree(delta_pos);
    hipFree(delta_vel);
}

__host__ void ParallelSimulator::reset() {
    // Consider using kernels to zero out memory 
    hipMemset(lambdas, 0, sizeof(double) * _n);
    hipMemset(densities, 0, sizeof(double) * _n);
    hipMemset(neighbor_starts, 0, sizeof(int) * _n)
    hipMemset(bins, 0, sizeof(int) * _total_cells);
    hipMemset(prefix_bins, 0, sizeof(int) * _total_cells);
    hipMemset(delta_pos, 0, sizeof(glm::dvec3) * _n);
    hipMemset(delta_vel, 0, sizeof(glm::dvec3) * _n);
}

__device__ double poly6(const glm::dvec3& r, const double h) {
    double r_mag = glm::length(r);
    if (r_mag > h) return 0.0;
    return 315.0 / (64 * Constants::pi * pow(h, 9)) * pow(h * h - r_mag * r_mag, 3);
}

__device__ glm::dvec3 grad_spiky(const glm::dvec3& r, const double h) {
    double r_mag = glm::length(r);
    if (r_mag > h) return glm::dvec3{0, 0, 0};
    return -45 / (Constants::pi * pow(h, 6) * std::max(r_mag, 1e-24)) * pow(h - r_mag, 2) * r;
}

__device__ glm::ivec3 get_cell_coords(glm::dvec3 pos) {
    ivec3 res {
        (pos.x - bbox_mins.x) / Constants::h,
        (pos.y - bbox_mins.y) / Constants::h,
        (pos.z - bbox_mins.z) / Constants::h
    };
    return clamp(res, ivec3{0, 0, 0}, ivec3{grid_width-1, grid_height-1, grid_length-1});
}

__device__ int get_cell_idx(glm::ivec3 coords) {
    return coords.x * grid_height * grid_length + coords.y * grid_length + coords.z;
    // return coords.x + coords.y * grid_width + coords.z * grid_width * grid_height;
} 

__global__ void compute_bins_kernel(Particle *particles, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    atomicAdd(&bins[cell_idx], 1);
}

__host__ void ParallelSimulator::compute_bins() {
    compute_bins_kernel<<<blocks, threads>>>(particles, bins, n);
    hipDeviceSynchronize();
}

__host__ void ParallelSimulator::compute_prefix_bins() {
    thrust::exclusive_scan(thrust::device, bins, bins + total_cells, prefix_bins);
    hipDeviceSynchronize();
}

__global__ void compute_sorted_particles_kernel(Particle *particles, int *prefix_bins, Particle *sorted_particles, n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    // Returns the value of prefix_bins[cell_idx] before the addition
    // THIS MIGHT BE BAD if atomic Add is not actually atomic 
    // cause multiple threads might get the same value of prefix_bins[cell_idx]
    int sorted_idx = atomicAdd(&prefix_bins[cell_idx], 1);
    sorted_particles[sorted_idx] = p;
}

__host__ void ParallelSimulator::compute_sorted_particles() {
    compute_sorted_particles_kernel<<<blocks, threads>>>(particles, prefix_bins, sorted_particles, n);
    hipDeviceSynchronize();
}

__global__ void compute_neighbor_starts_kernel(Particle *sorted_particles, int *neighbor_starts, int *prefix_bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    if (cell_idx == 0) {
        neighbor_starts[idx] = 0;
    } else {
        neighbor_starts[idx] = prefix_bins[cell_idx - 1];
    }
}

__host__ void ParallelSimulator::compute_neighbor_starts() {
    compute_neighbor_starts_kernel<<<blocks, threads>>>(sorted_particles, neighbor_starts, prefix_bins, n);
    hipDeviceSynchronize();
}

__host__ void ParallelSimulator::recompute_neighbors() {
    compute_bins();
    compute_prefix_bins();
    compute_sorted_particles();
    compute_neighbor_starts();
}

__device__ void compute_constraint(double *densities, int particle_id) {
    return densities[particle_id] / Constants::rest_density - 1.0;
}

__device__ glm::dvec3 compute_grad_constraint(Particle* sorted_particles, int neighbor_start, int neighbor_end, int constraint_id, int grad_id) {
    const Particle &constraint_particle = sorted_particles[constraint_id];
    const glm::dvec3& constraint_pos = constraint_particle.new_pos;
    if (constraint_id == grad_id) {
        glm::dvec3 res{0.0, 0.0, 0.0};
        for (int ni = neighbor_start; ni < neighbor_end; ni++) {
            const Particle &neighbor = sorted_particles[ni];
            const glm::dvec3& neighbor_pos = neighbor.new_pos;
            res += Constants::mass * grad_spiky(constraint_pos - neighbor_pos, Constants::h);
        }
        return res / Constants::rest_density;
    } else {
        return -Constants::mass * grad_spiky(constraint_pos - sorted_particles[grad_id].new_pos, Constants::h) / Constants::rest_density;
    }
}

__global__ void compute_lambdas_kernel(Particle *sorted_particles, double *densities, double *lambdas, int *neighbor_starts, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    double numerator = compute_constraint(densities, idx);
    double denominator = 0.0;
    int neighbor_start = neighbor_starts[idx];
    int neighbor_end = neighbor_start + bins[cell_idx];
    for (int ni = neighbor_start; ni < neighbor_end; ni++) {
        glm::dvec3 grad = compute_grad_constraint(sorted_particles, neighbor_start, neighbor_end, idx, ni);
        denominator += glm::dot(grad, grad) / Constants::mass;
    }
    denominator += Constants::epsilon;
    lambdas[idx] = -numerator / denominator;
}

__host__ void ParallelSimulator::compute_lambdas() {
    compute_lambdas_kernel<<<blocks, threads>>>(sorted_particles, densities, lambdas, neighbor_starts, prefix_bins, n);
    hipDeviceSynchronize();
}

__global__ void compute_delta_positions_kernel(Particle *sorted_particles, double *delta_pos, double *lambdas, int *neighbor_starts, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);

    double corr_q = poly6(Constants::corr_q * Constants::h * dvec3{1.0, 1.0, 1.0}, Constants::h);
    delta_pos[idx] = glm::dvec3{0.0, 0.0, 0.0};

    int neighbor_start = neighbor_starts[idx];
    int neighbor_end = neighbor_start + bins[cell_idx];
    for (int ni = neighbor_start; ni < neighbor_end; ni++) {
        const Particle &neighbor = sorted_particles[ni];
        double corr_kernel = poly6(p.new_pos - neighbor.new_pos, Constants::h); 
        double corr = -Constants::corr_k * std::pow(corr_kernel / corr_q, Constants::corr_n);
        glm::dvec3 grad_W = grad_spiky(p.new_pos - neighbor.new_pos, Constants::h);
        glm::dvec3 grad_W2 = compute_grad_constraint(sorted_particles, neighbor_start, neighbor_end, idx, ni);
        delta_pos[idx] += Constants::mass * (lambdas[idx] + lambdas[ni] + corr) * grad_W;
    }
    delta_pos[idx] *= (1.0 / Constants::mass) * (1.0 / Constants::rest_density);
}

__host__ void ParallelSimulator::compute_delta_positions() {
    compute_delta_positions<<<blocks, threads>>>(sorted_particles, delta_pos, lambdas, neighbor_starts, prefix_bins, n)
    hipDeviceSynchronize();
}


__global__ void update_positions_kernel(Particle *sorted_particles, glm::dvec3 *delta_pos, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    p.new_pos += delta_pos[idx];
}

__host__ void ParallelSimulator::update_positions() {
    update_positions_kernel(sorted_particles, delta_pos, n);
    hipDeviceSynchronize();
}

__global__ void update_collisions_kernel(Particle *sorted_particles, glm::dvec3 bbox_mins, glm::dvec3 bbox_maxs, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    for (int j = 0; j < 3; j++) {
        if (p.new_pos[j] < bbox_mins[j] + Constants::radius) {
            p.new_pos[j] = bbox_mins[j] + Constants::radius;
        } else if (p.new_pos[j] > bbox_maxs[j] - Constants::radius) {
            p.new_pos[j] = bbox_maxs[j] - Constants::radius;
        }
    }
}

__host__ void ParallelSimulator::update_collisions() {
    update_collisions_kernel(sorted_particles, bbox_mins, bbox_maxs, n);
    hipDeviceSynchronize();
}

__host__ void ParallelSimulator::simulate() {
    compute_densities(); // to use in lambdas and delta positions
    compute_lambdas();
    compute_delta_positions();
    update_positions();
    update_collisions();
}

__global__ void compute_velocities_kernel(Particle *particles, double elapsed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    p.vel += elapsed * Constants::g;
    p.new_pos = p.pos + elapsed * p.vel;
}

__host__ void ParallelSimulator::compute_velocities(double elapsed) {
    compute_velocities_kernel<<<blocks, threads>>>(particles, elapsed, n);
    hipDeviceSynchronize();
}

__global__ void compute_velocities_and_positions_kernel(Particle *sorted_particles, double elapsed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    p.vel = Constants::damping * (p.new_pos - p.pos) / elapsed;
    p.pos = p.new_pos;
}

__host__ void ParallelSimulator::compute_velocities_and_positions(double elapsed) {
    compute_velocities_and_positions_kernel<<<blocks, threads>>>(sorted_particles, elapsed, n);
    hipDeviceSynchronize();
}

__global__ void compute_densities_kernel(Particle *sorted_particles, double *densities, int *neighbor_starts, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    double density = 0.0;
    for (int ni = neighbor_starts[idx]; ni < neighbor_starts[idx] + bins[cell_idx]; ni++) {
        Particle &neighbor = sorted_particles[ni];
        density += Constant::mass * poly6(p.new_pos - neighbor.new_pos, Constants::h);
    }
    densities[idx] = density / Constants::rest_density - 1.0;
}

__host__ void ParallelSimulator::compute_densities() {
    compute_densities_kernel<<<blocks, threads>>>(sorted_particles, densities, neighbor_starts, bins, n);
    hipDeviceSynchronize();
}

__global__ void xsph_viscosity_kernel(Particle *sorted_particles, glm::dvec3 *delta_vel, double *densities, int *neighbor_starts, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    glm::dvec3 &delta_vel = delta_vel[idx];
    glm::ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    double delta_vel = 0.0;
    for (int ni = neighbor_starts[idx]; ni < neighbor_starts[idx] + bins[cell_idx]; ni++) {
        Particle &neighbor = sorted_particles[ni];
        glm::dvec3 vel = neighbor.vel - p.vel;
        double density = densities[ni];
        delta_vel += (Constants::mass / density) * poly6(p.new_pos - neighbor.new_pos, Constants::h);
    }
    delta_vel[idx] = delta_vel;
}

__host__ void ParallelSimulator::xsph_viscosity() {
    xsph_viscosity_kernel<<<blocks, threads>>>(sorted_particles, delta_vel, densities, neighbor_starts, bins, n);
    hipDeviceSynchronize();
}

__global__ void update_velocities_kernel(Particle *sorted_particles, glm::dvec3 *delta_vel, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = sorted_particles[idx];
    p.vel += delta_vel[idx];
}

__host__ void ParallelSimulator::update_velocities() {
    update_velocities_kernel<<<blocks, threads>>>(sorted_particles, delta_vel, n);
    hipDeviceSynchronize();
}

__host__ void ParallelSimulator::update(double elapsed, Scene& scene) {
    reset();

    // Copy scene particles from host to device memory 
    hipMemcpy(particles, scene.particles.data(), sizeof(Particle) * _n, hipMemcpyHostToDevice);

    // Initial forces 
    compute_velocities(elapsed);

    // Recompute neighbors 
    recompute_neighbors();

    // Simulation 
    for (int iter = 0; iter < Constants::solver_iterations; iter++) {
        simulate(); 
    }

    // Post processing (update velocities, positions, XSPH, vorticity)
    // TODO: vorticity
    compute_densities();
    compute_velocities_and_positions(elapsed);
    xsph_visocisty();
    update_velocities();

    // Copy particles back to host 
    hipMemcpy(scene.particles.data(), sorted_particles, sizeof(Particle) * _n, hipMemcpyDeviceToHost);
    return;
}