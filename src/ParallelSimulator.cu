#include "hip/hip_runtime.h"
#include "Simulator.hpp"
#include "Particle.hpp"
#include "Constants.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <GLFW/glfw3.h>
#include "Timer.hpp"
// #include <thrust/device_ptr.h>
// #include <thrust/device_malloc.h>
// #include <thrust/device_free.h>

Timer bigtimer;
Timer timer;
float iters = 0;
float initializationT=0, griddingT=0, densitiesT=0, lambdasT=0, deltasT=0, updatesT=0, postT=0, totalT=0;

#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {   
        fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

using namespace glm;

// consider moving Constants to here
struct GlobalConstants {
    dvec3 bbox_mins;
    dvec3 bbox_maxs; 
    size_t grid_width;
    size_t grid_height;
    size_t grid_length; 
    dvec3 g;
    double eps;
    double mass;
    double h;
    double radius;
    int solver_iterations;
    double pi;
    double rest_density;
    double corr_q;
    double corr_k;
    int corr_n;
    float xsph_c;
    double damping;
    int threads_per_block;
    long long MAX_NEIGHBORS;
};

__constant__ GlobalConstants GC; 

__host__ ParallelSimulator::ParallelSimulator(const Scene& scene) {
    _n = scene.particles.size();
    hipMalloc((void**)&n, sizeof(size_t));
    hipMemcpy((void**)&n, &_n, sizeof(size_t), hipMemcpyHostToDevice);   
    _blocks = (_n + Constants::threads_per_block - 1) / Constants::threads_per_block;
    _threads = Constants::threads_per_block;

    hipMalloc((void**)&bbox_mins, sizeof(dvec3));
    hipMalloc((void**)&bbox_maxs, sizeof(dvec3));
    hipMemcpy(&bbox_mins, &scene.bbox_mins, sizeof(dvec3), hipMemcpyHostToDevice);
    hipMemcpy(&bbox_maxs, &scene.bbox_maxs, sizeof(dvec3), hipMemcpyHostToDevice);

    hipMalloc((void**)&grid_width, sizeof(size_t));
    hipMalloc((void**)&grid_height, sizeof(size_t));
    hipMalloc((void**)&grid_length, sizeof(size_t));
    size_t width = (scene.bbox_maxs.x - scene.bbox_mins.x) / Constants::h + 1;
    size_t height = (scene.bbox_maxs.y - scene.bbox_mins.y) / Constants::h + 1;
    size_t length = (scene.bbox_maxs.z - scene.bbox_mins.z) / Constants::h + 1;
    hipMemcpy(&grid_width, &width, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&grid_height, &height, sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(&grid_length, &length, sizeof(size_t), hipMemcpyHostToDevice);

    hipMalloc((void**)&lambdas, sizeof(double) * _n);
    hipMalloc((void**)&densities, sizeof(double) * _n);
    hipMalloc((void**)&particles, sizeof(Particle) * _n);
    hipMalloc((void**)&neighbors, sizeof(int) * Constants::MAX_NEIGHBORS);
    hipMalloc((void**)&neighbor_starts, sizeof(int) * _n);
    hipMalloc((void**)&neighbor_sizes, sizeof(int) * _n);

    _total_cells = width * height * length; // overflow?
    hipMalloc((void**)&total_cells, sizeof(size_t));
    hipMemcpy(&total_cells, &_total_cells, sizeof(size_t), hipMemcpyHostToDevice); 
    hipMalloc((void**)&bins, sizeof(int) * _total_cells);
    hipMalloc((void**)&prefix_bins, sizeof(int) * _total_cells);
    hipMalloc((void**)&grid_starts, sizeof(int) * _total_cells);
    hipMalloc((void**)&grid, sizeof(int) * _n);

    hipMalloc((void**)&delta_pos, sizeof(dvec3) * _n);
    hipMalloc((void**)&delta_vel, sizeof(dvec3) * _n);

    GlobalConstants _GC;
    _GC.bbox_mins = scene.bbox_mins;
    _GC.bbox_maxs = scene.bbox_maxs;
    _GC.grid_width = width;
    _GC.grid_height = height;
    _GC.grid_length = length;
    _GC.g = Constants::g;
    _GC.eps = Constants::eps;
    _GC.mass = Constants::mass;
    _GC.h = Constants::h;
    _GC.radius = Constants::radius;
    _GC.solver_iterations = Constants::solver_iterations;
    _GC.pi = Constants::pi;
    _GC.rest_density = Constants::rest_density;
    _GC.corr_q = Constants::corr_q;
    _GC.corr_k = Constants::corr_k;
    _GC.corr_n = Constants::corr_n;
    _GC.xsph_c = Constants::xsph_c;
    _GC.damping = Constants::damping;
    _GC.threads_per_block = Constants::threads_per_block;
    _GC.MAX_NEIGHBORS = Constants::MAX_NEIGHBORS;

    hipMemcpyToSymbol(HIP_SYMBOL(GC), &_GC, sizeof(GlobalConstants));
}

__host__ ParallelSimulator::~ParallelSimulator() {
    hipFree(&n);
    hipFree(&total_cells);
    hipFree(&bbox_mins);
    hipFree(&bbox_maxs);
    hipFree(&grid_width);
    hipFree(&grid_height);
    hipFree(&grid_length);
    hipFree(lambdas);
    hipFree(densities);
    hipFree(particles);
    hipFree(neighbors);
    hipFree(bins);
    hipFree(prefix_bins);
    hipFree(delta_pos);
    hipFree(delta_vel);
    hipFree(neighbor_sizes);
    hipFree(neighbor_starts);
    hipFree(grid);
}

__host__ void ParallelSimulator::reset() {
    // Consider using kernels to zero out memory 
    hipMemset(lambdas, 0, sizeof(double) * _n);
    hipMemset(densities, 0, sizeof(double) * _n);
    hipMemset(neighbor_sizes, 0, sizeof(int) * _n);
    hipMemset(bins, 0, sizeof(int) * _total_cells);
    hipMemset(prefix_bins, 0, sizeof(int) * _total_cells);
    hipMemset(delta_pos, 0, sizeof(dvec3) * _n);
    hipMemset(delta_vel, 0, sizeof(dvec3) * _n);
}

__device__ __forceinline__ double poly6(const dvec3& r, const double h) {
    double r_mag = length(r);
    double powh3 = h*h*h;
    double powh9 = powh3*powh3*powh3;
    double h_rmag = h*h - r_mag*r_mag;
    double h_rmag3 = h_rmag*h_rmag*h_rmag;
    return r_mag > h ? 0.0 : 315.0 / (64 * GC.pi * powh9) * h_rmag3;
}

__device__ __forceinline__ dvec3 grad_spiky(const dvec3& r, const double h) {
    double r_mag = length(r);
    dvec3 zero(0);
    double powh2 = h*h;
    double powh6 = powh2*powh2*powh2;
    double dist2 = dot(h-r_mag, h-r_mag);
    return r_mag > h ? zero : -45 / (GC.pi * powh6 * max(r_mag, 1e-24)) * dist2 * r;
}

__device__ ivec3 get_cell_coords(dvec3 pos) {
    ivec3 res {
        (pos.x - GC.bbox_mins.x) / GC.h,
        (pos.y - GC.bbox_mins.y) / GC.h,
        (pos.z - GC.bbox_mins.z) / GC.h
    };
    return clamp(res, ivec3{0, 0, 0}, ivec3{GC.grid_width-1, GC.grid_height-1, GC.grid_length-1});
}

__device__ int get_cell_idx(ivec3 coords) {
    return coords.x * GC.grid_height * GC.grid_length + coords.y * GC.grid_length + coords.z;
} 

__global__ void compute_bins_kernel(Particle *particles, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    atomicAdd(&bins[cell_idx], 1);
}

__global__ void compute_sorted_grid_kernel(Particle *particles, int *prefix_bins, int *grid, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    int sorted_idx = atomicAdd(&prefix_bins[cell_idx], 1);
    grid[sorted_idx] = p.id;
}

__global__ void compute_grid_starts_kernel(Particle *particles, int *grid_starts, int *prefix_bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    if (cell_idx == 0) {
        grid_starts[cell_idx] = 0;
    } else {
        grid_starts[cell_idx] = prefix_bins[cell_idx - 1];
    }
}


__host__ void ParallelSimulator::recompute_grid() {
    compute_bins_kernel<<<_blocks, _threads>>>(particles, bins, _n);
    // cudaCheckError(hipDeviceSynchronize());
    thrust::exclusive_scan(thrust::device, bins, bins + _total_cells, prefix_bins);
    // cudaCheckError(hipDeviceSynchronize());
    compute_sorted_grid_kernel<<<_blocks, _threads>>>(particles, prefix_bins, grid, _n);
    // cudaCheckError(hipDeviceSynchronize());
    compute_grid_starts_kernel<<<_blocks, _threads>>>(particles, grid_starts, prefix_bins, _n);
    // cudaCheckError(hipDeviceSynchronize());
}

__global__ void compute_neighbor_sizes_kernel(Particle *particles, int *neighbor_sizes, int *bins, int _n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= _n) return;
    Particle &p = particles[idx];
    ivec3 our_coords = get_cell_coords(p.new_pos);

    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dz = -1; dz <= 1; ++dz) {
                ivec3 coords = our_coords + ivec3{dx, dy, dz};
                if (coords.x < 0 || coords.x >= GC.grid_width ||
                    coords.y < 0 || coords.y >= GC.grid_height ||
                    coords.z < 0 || coords.z >= GC.grid_length) continue;
                
                int cell_idx = get_cell_idx(coords);
                neighbor_sizes[idx] += bins[cell_idx];
            }
        }
    }
}


__global__ void compute_neighbors_kernel(Particle *particles, int *neighbors, int *neighbor_sizes, int *neighbor_starts, int *grid, int *grid_starts, int *bins, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 our_coords = get_cell_coords(p.new_pos);
    // Hacky way to make ourself always the first neighbor
    neighbors[neighbor_starts[idx]] = idx;
    int current_neighbor = 1;

    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dz = -1; dz <= 1; ++dz) {
                ivec3 coords = our_coords + ivec3{dx, dy, dz};
                if (coords.x < 0 || coords.x >= GC.grid_width ||
                    coords.y < 0 || coords.y >= GC.grid_height ||
                    coords.z < 0 || coords.z >= GC.grid_length) continue;
                
                int cell_idx = get_cell_idx(coords);
                for (int j = grid_starts[cell_idx]; j < grid_starts[cell_idx] + bins[cell_idx]; ++j) {
                    if (grid[j] != idx) {
                        int nidx = neighbor_starts[idx] + current_neighbor;
                        if (nidx < GC.MAX_NEIGHBORS) {
                            neighbors[nidx] = grid[j];
                            current_neighbor++;
                        } else {
                            printf("WARNING: MAX_NEIGHBORS exceeded\n");
                            return;
                        }
                    }
                }
            }
        }
    }
}

__host__ void ParallelSimulator::recompute_neighbors() {
    compute_neighbor_sizes_kernel<<<_blocks, _threads>>>(particles, neighbor_sizes, bins, _n);
    // cudaCheckError(hipDeviceSynchronize());
    thrust::exclusive_scan(thrust::device, neighbor_sizes, neighbor_sizes + _n, neighbor_starts);
    // cudaCheckError(hipDeviceSynchronize());
    compute_neighbors_kernel<<<_blocks, _threads>>>(particles, neighbors, neighbor_sizes, neighbor_starts, grid, grid_starts, bins, _n);
    fflush(stdout);
    // cudaCheckError(hipDeviceSynchronize());
}

__device__ double compute_constraint(double *densities, int particle_id) {
    return densities[particle_id] / GC.rest_density - 1.0;
}

__device__ dvec3 compute_grad_constraint(Particle* particles, int constraint_id, int grad_id) {
    const Particle &constraint_particle = particles[constraint_id];
    const dvec3& constraint_pos = constraint_particle.new_pos;
    return -GC.mass * grad_spiky(constraint_pos - particles[grad_id].new_pos, GC.h) / GC.rest_density;
}

__device__ dvec3 compute_grad_constraint_self(Particle* particles, int* neighbors, int neighbor_start, int neighbor_end, int constraint_id) {
    const Particle &constraint_particle = particles[constraint_id];
    const dvec3& constraint_pos = constraint_particle.new_pos;
    dvec3 res{0.0, 0.0, 0.0};
    for (int ni = neighbor_start; ni < neighbor_end; ni++) {
        int neighbor_id = neighbors[ni];
        Particle& neighbor = particles[neighbor_id];
        const dvec3& neighbor_pos = neighbor.new_pos;
        res += GC.mass * grad_spiky(constraint_pos - neighbor_pos, GC.h);
    }
    return res / GC.rest_density;
}

__global__ void compute_lambdas_kernel(Particle *particles, int *neighbors, double *densities, double *lambdas, int *neighbor_starts, int *neighbor_sizes, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    int neighbor_start = neighbor_starts[idx];
    int neighbor_end = neighbor_start + neighbor_sizes[idx];
    double numerator = compute_constraint(densities, idx);
    dvec3 grad_self  = compute_grad_constraint_self(particles, neighbors, neighbor_start, neighbor_end, idx);
    double denominator = dot(grad_self, grad_self) / GC.mass;
    for (int ni = neighbor_start + 1; ni < neighbor_end; ni++) {
        int neighbor_id = neighbors[ni];
        Particle& neighbor = particles[neighbor_id];
        dvec3 grad = compute_grad_constraint(particles, idx, neighbor.id);
        denominator += dot(grad, grad) / GC.mass;
    }
    denominator += GC.eps;
    lambdas[idx] = -numerator / denominator;
}

__host__ void ParallelSimulator::compute_lambdas() {
    compute_lambdas_kernel<<<_blocks, _threads>>>(particles, neighbors, densities, lambdas, neighbor_starts, neighbor_sizes, _n);
    // cudaCheckError(hipDeviceSynchronize());
}

__global__ void compute_delta_positions_kernel(Particle *particles, int *neighbors, dvec3 *delta_pos, double *lambdas, int *neighbor_starts, int *neighbor_sizes, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);

    double corr_q = poly6(GC.corr_q * GC.h * dvec3{1.0, 0.0, 0.0}, GC.h);
    delta_pos[idx] = dvec3{0.0, 0.0, 0.0};

    int neighbor_start = neighbor_starts[idx];
    int neighbor_end = neighbor_start + neighbor_sizes[idx];
    for (int ni = neighbor_start; ni < neighbor_end; ni++) {
        int neighbor_id = neighbors[ni];
        Particle &neighbor = particles[neighbor_id];
        double corr_kernel = poly6(p.new_pos - neighbor.new_pos, GC.h); 
        double ratio = corr_kernel / corr_q;
        double ratio2 = ratio*ratio;
        double ratio4 = ratio2*ratio2;
        double corr = -GC.corr_k * ratio4;
        dvec3 grad_W = grad_spiky(p.new_pos - neighbor.new_pos, GC.h);
        delta_pos[idx] += GC.mass * (lambdas[idx] + lambdas[neighbor.id] + corr) * grad_W;
    }
    delta_pos[idx] *= (1.0 / GC.mass) * (1.0 / GC.rest_density);
}


__host__ void ParallelSimulator::compute_delta_positions() {
    compute_delta_positions_kernel<<<_blocks, _threads>>>(particles, neighbors, delta_pos, lambdas, neighbor_starts, neighbor_sizes, _n);
    // cudaCheckError(hipDeviceSynchronize());
}


__global__ void update_positions_kernel(Particle *particles, dvec3 *delta_pos, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    p.new_pos += delta_pos[idx];
}

__host__ void ParallelSimulator::update_positions() {
    update_positions_kernel<<<_blocks, _threads>>>(particles, delta_pos, _n);
    // cudaCheckError(hipDeviceSynchronize());
}

__global__ void update_collisions_kernel(Particle *particles, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    for (int j = 0; j < 3; j++) {
        if (p.new_pos[j] < GC.bbox_mins[j] + GC.radius) {
            p.new_pos[j] = GC.bbox_mins[j] + GC.radius;
        } else if (p.new_pos[j] > GC.bbox_maxs[j] - GC.radius) {
            p.new_pos[j] = GC.bbox_maxs[j] - GC.radius;
        }
    }
}

__host__ void ParallelSimulator::update_collisions() {
    update_collisions_kernel<<<_blocks, _threads>>>(particles, _n);
    // cudaCheckError(hipDeviceSynchronize());
}

__host__ void ParallelSimulator::simulate() {
    hipDeviceSynchronize();
    timer.start();
    compute_densities(); // to use in lambdas and delta positions
    hipDeviceSynchronize();
    densitiesT += timer.stop();
    compute_lambdas();
    hipDeviceSynchronize();
    lambdasT += timer.stop();
    compute_delta_positions();
    hipDeviceSynchronize();
    deltasT += timer.stop();
    update_positions();
    update_collisions();
    hipDeviceSynchronize();
    updatesT += timer.stop();
}

__global__ void compute_velocities_kernel(Particle *particles, double elapsed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    p.vel += elapsed * GC.g;
    p.new_pos = p.pos + elapsed * p.vel;
}

__host__ void ParallelSimulator::compute_velocities(double elapsed) {
    compute_velocities_kernel<<<_blocks, _threads>>>(particles, elapsed, _n);
}

__global__ void compute_velocities_and_positions_kernel(Particle *particles, double elapsed, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    p.vel = GC.damping * (p.new_pos - p.pos) / elapsed;
    p.pos = p.new_pos;
}

__host__ void ParallelSimulator::compute_velocities_and_positions(double elapsed) {
    compute_velocities_and_positions_kernel<<<_blocks, _threads>>>(particles, elapsed, _n);
}

__global__ void compute_densities_kernel(Particle *particles, int *neighbors, double *densities, int *neighbor_starts, int *neighbor_sizes, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    double density = 0.0;
    for (int ni = neighbor_starts[idx]; ni < neighbor_starts[idx] + neighbor_sizes[idx]; ni++) {
        int neighbor_id = neighbors[ni];
        Particle &neighbor = particles[neighbor_id];
        density += GC.mass * poly6(p.new_pos - neighbor.new_pos, GC.h);
    }
    densities[idx] = density;
}

__host__ void ParallelSimulator::compute_densities() {
    compute_densities_kernel<<<_blocks, _threads>>>(particles, neighbors, densities, neighbor_starts, neighbor_sizes, _n);
}

__global__ void xsph_viscosity_kernel(Particle *particles, int *neighbors, dvec3 *delta_vel, double *densities, int *neighbor_starts, int *neighbor_sizes, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    ivec3 coords = get_cell_coords(p.new_pos);
    int cell_idx = get_cell_idx(coords);
    delta_vel[idx] = dvec3{0.0, 0.0, 0.0};
    for (int ni = neighbor_starts[idx]; ni < neighbor_starts[idx] + neighbor_sizes[idx]; ni++) {
        int neighbor_id = neighbors[ni];
        Particle &neighbor = particles[neighbor_id];
        dvec3 vel = neighbor.vel - p.vel;
        double density = densities[neighbor.id];
        delta_vel[idx] += (GC.mass / density) * vel * poly6(p.new_pos - neighbor.new_pos, GC.h);
    }
}

__host__ void ParallelSimulator::xsph_viscosity() {
    xsph_viscosity_kernel<<<_blocks, _threads>>>(particles, neighbors, delta_vel, densities, neighbor_starts, neighbor_sizes, _n);
}

__global__ void update_velocities_kernel(Particle *particles, dvec3 *delta_vel, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;
    Particle &p = particles[idx];
    p.vel += delta_vel[idx];
}

__host__ void ParallelSimulator::update_velocities() {
    update_velocities_kernel<<<_blocks, _threads>>>(particles, delta_vel, _n);
}

__host__ void ParallelSimulator::update(double elapsed, Scene& scene) {
    iters++;
    hipDeviceSynchronize();
    bigtimer.start();
    timer.start();
    reset();

    // Copy scene particles from host to device memory 
    hipMemcpy(particles, scene.particles.data(), sizeof(Particle) * _n, hipMemcpyHostToDevice);

    // Initial forces 
    compute_velocities(elapsed);
    hipDeviceSynchronize();
    initializationT += timer.stop();

    // Recompute neighbors 
    recompute_grid();
    recompute_neighbors();

    hipDeviceSynchronize();
    griddingT += timer.stop();

    // Simulation 
    for (int iter = 0; iter < Constants::solver_iterations; iter++) {
        simulate(); 
    }

    // Post processing (update velocities, positions, XSPH, vorticity)
    // TODO: vorticity
    hipDeviceSynchronize();
    timer.start();
    compute_densities();
    compute_velocities_and_positions(elapsed);
    xsph_viscosity();
    update_velocities();

    // Copy particles back to host 
    hipMemcpy(scene.particles.data(), particles, sizeof(Particle) * _n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    postT += timer.stop();
    totalT += bigtimer.stop();

    // cout << "Initialization: " << initializationT / iters << endl;
    // cout << "Gridding/neighbors: " << griddingT / iters << endl;
    // cout << "Densities: " << densitiesT / iters << endl;
    // cout << "Lambdas: " << lambdasT / iters << endl;
    // cout << "Deltas: " << deltasT / iters << endl;
    // cout << "Position/velocity updates: " << updatesT / iters << endl;
    // cout << "Post-processing: " << postT / iters << endl;
    // cout << "Everything: " << totalT / iters << endl;
    // cout << endl;

    cout << initializationT / iters << endl;
    cout << griddingT / iters << endl;
    cout << densitiesT / iters << endl;
    cout << lambdasT / iters << endl;
    cout << deltasT / iters << endl;
    cout << updatesT / iters << endl;
    cout << postT / iters << endl;
    cout << totalT / iters << endl;
    cout << endl;
    return;
}