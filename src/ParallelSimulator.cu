#include "Simulator.hpp"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

ParallelSimulator::ParallelSimulator(const Scene& scene) {


}

ParallelSimulator::~ParallelSimulator() {

}

void ParallelSimulator::update(double elapsed, Scene& scene) {
    return;
}

glm::ivec3 ParallelSimulator::get_cell_coords(glm::dvec3 pos) {
    return glm::ivec3{0, 0, 0};
}

__device__ __host__ int ParallelSimulator::get_cell_idx(glm::ivec3 coords) {
    return 0;
}
